
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK_ERROR(err) {                                               \
    if (err != hipSuccess) {                                                 \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;  \
        exit(EXIT_FAILURE);                                                   \
    }                                                                         \
}

template<class T>
struct MyArray {
    T* data;
    int elementCount;
};

template<class T>
struct MyStruct {
    MyArray<T> arrayA;
    MyArray<T> arrayB;
};

template<class T>
__global__ void SumArrays(MyStruct<T> inStruct, MyArray<T> outArray){
    outArray.data[threadIdx.x] = inStruct.arrayA.data[threadIdx.x] + inStruct.arrayB.data[threadIdx.x];
}

int main() {
    constexpr int size = 3;

    // Declare and initialize struct on the host
    MyStruct<float> h_myStruct;

    h_myStruct.arrayA.elementCount = size;
    h_myStruct.arrayA.data = new float[size];
    h_myStruct.arrayA.data[0] = 1.0f;
    h_myStruct.arrayA.data[1] = 2.0f;
    h_myStruct.arrayA.data[2] = 3.0f;

    h_myStruct.arrayB.elementCount = size;
    h_myStruct.arrayB.data = new float[size];
    h_myStruct.arrayB.data[0] = 4.0f;
    h_myStruct.arrayB.data[1] = 5.0f;
    h_myStruct.arrayB.data[2] = 6.0f;

    // Declare struct and malloc memory on the device
    MyStruct<float> d_myStruct;
    CUDA_CHECK_ERROR(hipMalloc(&(d_myStruct.arrayA.data), h_myStruct.arrayA.elementCount * sizeof(h_myStruct.arrayA.data[0])));
    CUDA_CHECK_ERROR(hipMalloc(&(d_myStruct.arrayB.data), h_myStruct.arrayB.elementCount * sizeof(h_myStruct.arrayB.data[0])));

    // Copy the host struct to the device
    d_myStruct.arrayA.elementCount = h_myStruct.arrayA.elementCount;
    CUDA_CHECK_ERROR(hipMemcpy(d_myStruct.arrayA.data, h_myStruct.arrayA.data, h_myStruct.arrayA.elementCount * sizeof(h_myStruct.arrayA.data[0]), hipMemcpyHostToDevice));
    d_myStruct.arrayB.elementCount = h_myStruct.arrayB.elementCount;
    CUDA_CHECK_ERROR(hipMemcpy(d_myStruct.arrayB.data, h_myStruct.arrayB.data, h_myStruct.arrayB.elementCount * sizeof(h_myStruct.arrayB.data[0]), hipMemcpyHostToDevice));

    // Declare struct and malloc memory on the device to hold the result
    MyArray<float> d_out;
    CUDA_CHECK_ERROR(hipMalloc(&(d_out.data), h_myStruct.arrayA.elementCount * sizeof(h_myStruct.arrayA.data[0])));

    // Spawn the kernel to sum the arrays in the struct and store it in d_out
    SumArrays<float><<<1,size>>>(d_myStruct, d_out);
    CUDA_CHECK_ERROR(hipDeviceSynchronize());

    // Allocate output array on the host and copy the result back
    MyArray<float> h_out;
    h_out.elementCount = size;
    h_out.data = new float[size];

    // Copy the device array to the host
    CUDA_CHECK_ERROR(hipMemcpy(h_out.data, d_out.data, h_out.elementCount * sizeof(h_out.data[0]), hipMemcpyDeviceToHost));
    
    for (int i = 0; i < h_out.elementCount; ++i) {
        std::cout << h_out.data[i] << std::endl;
    }

    // Free host memory
    delete[] h_myStruct.arrayA.data;
    delete[] h_myStruct.arrayB.data;
    delete[] h_out.data;

    // Free device memory
    CUDA_CHECK_ERROR(hipFree(d_myStruct.arrayA.data));
    CUDA_CHECK_ERROR(hipFree(d_myStruct.arrayB.data));
    CUDA_CHECK_ERROR(hipFree(d_out.data));
}